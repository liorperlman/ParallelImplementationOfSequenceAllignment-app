#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include "myMacro.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>


__device__ SignType defineSignCuda(char c1, char c2);
__device__ char *myStrchr(const char *s, int c);
__device__ char getCharReplacementCuda(char c1, char c2, int modValue, SignType minimum);


__global__ void createMutantCuda(SignType *d_signs, char *d_seq1, char *d_seq2, int numElements, char *d_mutant, int modValue, SignType minimum){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements){
		//if (d_seq2[i] == letterToReplace){
		if (modValue == 1)
			d_mutant[i] = getCharReplacementCuda(d_seq1[i], d_seq2[i], modValue, minimum);
		else{ //minimum
			if (defineSignCuda(d_seq1[i], d_seq2[i]) != minimum)	
				d_mutant[i] = getCharReplacementCuda(d_seq1[i], d_seq2[i], modValue, minimum);
			else
				d_mutant[i] = d_seq2[i];
		}
		
		d_signs[i] = defineSignCuda(d_seq1[i], d_mutant[i]);
	}		
}

__device__ char getCharReplacementCuda(char c1, char c2, int modValue, SignType minimum){
	int i;
	char *ptr1, *ptr2;
	SignType sign, sign1, sign2;
	const char *similar[SIM_SIZE] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
	int similarSize = SIM_SIZE;
	
	if (modValue == 1) {	// maximum
		// Check if both c1 and c2 are in one of Similar groups
		for (i = 0; i < similarSize; i++) {
			ptr1 = myStrchr(similar[i], c1);
			ptr2 = myStrchr(similar[i], c2);
			if (ptr1 != NULL && ptr2 != NULL)
				return c2;
		}
		// If not we can replace it with the char from seq1 to increase the score
		return c1;
	}
	else {	// modValue = 0 => minimum
		// Find a char who's giving the worst score in comparison with c1 and not similar in comparison with c2 
		sign = defineSignCuda(c1, c2);
		if (sign != minimum){
			for (i = 0; i < 26; i++){
				sign1 = defineSignCuda(c1, i+65);
				sign2 = defineSignCuda(c2, i+65);
				if (sign1 == minimum && sign2 != Similar)
					return i+65;	
			}
			return c2;
		}
		else{ // sign = minimum
			return c2;
		}
	}
}

__device__ SignType defineSignCuda(char c1, char c2) {
	int i;
	char *ptr1, *ptr2;
	const char *similar[SIM_SIZE] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
	int similarSize = SIM_SIZE;
	const char *almostSimilar[ALMOST_SIM_SIZE] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK",
							 			"NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };
	int almostSimSize = ALMOST_SIM_SIZE;
	
	// Check if Equal
	if (c1 == c2)
		return Equal;

	// Check if both c1 and c2 are in one of Similar groups
	for (i = 0; i < similarSize; i++) {
		ptr1 = myStrchr(similar[i], c1);
		ptr2 = myStrchr(similar[i], c2);
		if (ptr1 != NULL && ptr2 != NULL)
			return Similar;
	}

	// Check if both c1 and c2 are in one of Almost Similar groups
	for (i = 0; i < almostSimSize; i++) {
		ptr1 = myStrchr(almostSimilar[i], c1);
		ptr2 = myStrchr(almostSimilar[i], c2);
		if (ptr1 != NULL && ptr2 != NULL)
			return AlmostSimilar;
	}
	// Not Equal and Not found in Similar or AlmostSimilar groups 
	return NotEqual;
}

__device__ char *myStrchr(const char *s, int c)
{
    while (*s != (char)c)
        if (!*s++)
            return NULL;
    return (char *)s;
}

int computeOnGPU(SignType *h_signs, char *h_seq1, char *h_seq2, int numElements, char *h_mutant, int modValue, SignType minimum) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t charSize = numElements * sizeof(char);
    size_t signSize = numElements * sizeof(SignType);

    // Allocate memory on GPU to copy the data from the host
    char *d_seq1;
    char *d_seq2;
    char *d_mutant;
    SignType *d_signs;
    err = hipMalloc((void **)&d_seq1, charSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for seq1 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    } 
    err = hipMalloc((void **)&d_seq2, charSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_mutant, charSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_signs, signSize);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for signs - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_seq1, h_seq1, charSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device for seq1 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_seq2, h_seq2, charSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    createMutantCuda<<<blocksPerGrid, threadsPerBlock>>>(d_signs, d_seq1, d_seq2, numElements, d_mutant, modValue, minimum);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the result from GPU to the host memory.
    err = hipMemcpy(h_mutant, d_mutant, charSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(h_signs, d_signs, signSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_seq1) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_seq2) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_mutant) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	if (hipFree(d_signs) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
